#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }


        __global__ void upsweep(int* data, int twod, int twod1, int n) {
            int index = threadIdx.x + blockIdx.x * blockDim.x;
            int k = index * twod1;
            if (k + twod1 - 1 < n) {
                data[k + twod1 - 1] += data[k + twod - 1];
            }
        }


        __global__ void downsweep(int* data, int twod, int twod1, int n) {
            int index = threadIdx.x + blockIdx.x * blockDim.x;
            int k = index * twod1;
            if (k + twod1 - 1 < n) {
                int t = data[k + twod - 1];
                data[k + twod - 1] = data[k + twod1 - 1];
                data[k + twod1 - 1] += t;
            }
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */

         // Internal version that controls timing
        void scan_internal(int n, int* odata, const int* idata, bool timing) {
            if (timing) timer().startGpuTimer();

            int pow2Len = 1 << ilog2ceil(n);
            int* dev_data;
            hipMalloc(&dev_data, pow2Len * sizeof(int));
            hipMemset(dev_data, 0, pow2Len * sizeof(int));
            hipMemcpy(dev_data, idata, n * sizeof(int), hipMemcpyHostToDevice);

            int blockSize = 512;
            for (int d = 1; d <= ilog2ceil(pow2Len); ++d) {
                int twod = 1 << (d - 1);
                int twod1 = 1 << d;
                int numThreads = pow2Len / twod1;
                int numBlocks = (numThreads + blockSize - 1) / blockSize;
                upsweep << <numBlocks, blockSize >> > (dev_data, twod, twod1, pow2Len);
                hipDeviceSynchronize();
            }

            hipMemset(dev_data + pow2Len - 1, 0, sizeof(int)); // zero for exclusive

            for (int d = ilog2ceil(pow2Len); d >= 1; --d) {
                int twod = 1 << (d - 1);
                int twod1 = 1 << d;
                int numThreads = pow2Len / twod1;
                int numBlocks = (numThreads + blockSize - 1) / blockSize;
                downsweep << <numBlocks, blockSize >> > (dev_data, twod, twod1, pow2Len);
                hipDeviceSynchronize();
            }

            hipMemcpy(odata, dev_data, n * sizeof(int), hipMemcpyDeviceToHost);
            hipFree(dev_data);

            if (timing) timer().endGpuTimer();
        }

        void scan(int n, int* odata, const int* idata) {
            scan_internal(n, odata, idata, true);
        }



        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int* odata, const int* idata) {
            int* dev_idata, * dev_bools, * dev_indices, * dev_odata;

            int pow2n = 1 << ilog2ceil(n);
            hipMalloc(&dev_idata, pow2n * sizeof(int));
            hipMalloc(&dev_bools, pow2n * sizeof(int));
            hipMalloc(&dev_indices, pow2n * sizeof(int));
            hipMalloc(&dev_odata, pow2n * sizeof(int));
            hipMemset(dev_odata, 0, pow2n * sizeof(int));

            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);

            timer().startGpuTimer();

            int blockSize = 128;
            int numBlocks = (n + blockSize - 1) / blockSize;

            StreamCompaction::Common::kernMapToBoolean<<<numBlocks, blockSize >>>(
                n, dev_bools, dev_idata);
            hipDeviceSynchronize();

            scan_internal(n, dev_indices, dev_bools, false);
            hipDeviceSynchronize();

            StreamCompaction::Common::kernScatter<<<numBlocks, blockSize >>> (
                n, dev_odata, dev_idata, dev_bools, dev_indices);
            hipDeviceSynchronize();

            timer().endGpuTimer();

            int count;
            hipMemcpy(&count, dev_indices + n - 1, sizeof(int), hipMemcpyDeviceToHost);

            int lastBool;
            hipMemcpy(&lastBool, dev_bools + n - 1, sizeof(int), hipMemcpyDeviceToHost);
            count += lastBool;

            hipMemcpy(odata, dev_odata, count * sizeof(int), hipMemcpyDeviceToHost);

            hipFree(dev_idata);
            hipFree(dev_bools);
            hipFree(dev_indices);
            hipFree(dev_odata);

            return count;
        }

    }
}