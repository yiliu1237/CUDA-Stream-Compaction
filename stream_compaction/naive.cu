#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

#include <iostream>

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        //naive inclusive scan
        __global__ void naiveScanStep(int n, int d, const int* input, int* output) {
            int k = threadIdx.x + blockIdx.x * blockDim.x;
            if (k >= n) return;

            if (k >= (1 << (d - 1))) {
                output[k] = input[k - (1 << (d - 1))] + input[k];
            }
            else {
                output[k] = input[k];
            }
        }

        void scan(int n, int* odata, const int* idata) {

            int* dev_ping;
            int* dev_pong;

            hipMalloc(&dev_ping, n * sizeof(int));
            hipMalloc(&dev_pong, n * sizeof(int));

            hipMemcpy(dev_ping, idata, n * sizeof(int), hipMemcpyHostToDevice);

            int blockSize = 1024;
            int numBlocks = (n + blockSize - 1) / blockSize;

            int depth = ilog2ceil(n);

            timer().startGpuTimer();
            for (int d = 1; d <= depth; d++) {
                naiveScanStep << <numBlocks, blockSize >> > (n, d, dev_ping, dev_pong);
                //hipDeviceSynchronize();

                // Swap buffers
                std::swap(dev_ping, dev_pong);
            }
            timer().endGpuTimer();

            // dev_ping now has the inclusive scan result
            // Convert to exclusive scan
            hipMemcpy(odata + 1, dev_ping, (n - 1) * sizeof(int), hipMemcpyDeviceToHost);
            odata[0] = 0;

            hipFree(dev_ping);
            hipFree(dev_pong);

        }
    }
}
